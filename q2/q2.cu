#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>

__global__ void out(int* a, int wa) {
	int ridA = threadIdx.x;
	int ha = blockDim.x;

	int p;
	if (ridA > 0 && ridA < ha - 1) {
		for (int k = 1; k < wa - 1; k++) {
			p = a[ridA * wa + k];
			int rem = 0, bin = 0, place = 1;

			while (p != 0) {
				rem = p % 2;
				p = p / 2;
				bin = bin + rem * place;
				place = place * 10;
			}

			a[ridA * wa + k] = bin;
		}
	}
}

int main() {
	int* a, ha, wa;
	int* da;

	printf("Enter the no.of rows and columns: ");
	scanf("%d %d", &ha, &wa);

	int size = sizeof(int) * ha * wa;

	a = (int*)malloc(size);

	printf("Enter the matrix: ");
	for (int i = 0; i < ha; i++)
		for (int j = 0; j < wa; j++)
			scanf("%d", &a[i * wa + j]);

	hipMalloc((void**)&da, size);

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);

	out << <1, ha >> > (da, wa);

	hipMemcpy(a, da, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < ha; i++) {
		for (int j = 0; j < wa; j++)
			printf("%d\t", a[i * wa + j]);
		printf("\n");
	}
	hipFree(da);

	return 0;
}