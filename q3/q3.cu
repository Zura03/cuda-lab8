#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>

#define NNZ 7
#define N 4
#define BLOCK_SIZE 4

__global__ void spmv(float* data, int* col_index, int* row_ptr, float* x, float* y) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < N) {
		float dot = 0;
		int row_start = row_ptr[row];
		int row_end = row_ptr[row + 1];
		for (int elem = row_start; elem < row_end; elem++) {
			dot += data[elem] * x[col_index[elem]];
		}
		y[row] = dot;
	}
}

int main() {
	float data[NNZ] = { 3.0, 1.0, 2.0, 4.0, 1.0, 1.0, 1.0 };
	int col_index[NNZ] = { 0,2,1,2,3,0,3 };
	int row_pointers[N + 1] = { 0,2,2,5,7 };

	float x[N] = { 1.0, 2.0, 3.0, 4.0 };

	float* d_y;
	hipMalloc((void**)&d_y, N * sizeof(float));

	float* d_data, * d_x;
	int* d_col_index, * d_row_ptr;
	hipMalloc((void**)&d_data, NNZ * sizeof(float));
	hipMalloc((void**)&d_col_index, NNZ * sizeof(int));
	hipMalloc((void**)&d_row_ptr, (N+1) * sizeof(int));
	hipMalloc((void**)&d_x, N * sizeof(float));

	hipMemcpy(d_data, data, NNZ * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_col_index, col_index, NNZ * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr, row_pointers, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

	int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
	spmv << <num_blocks, BLOCK_SIZE >> > (d_data, d_col_index, d_row_ptr, d_x, d_y);

	float y[N];
	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_data);
	hipFree(d_col_index);
	hipFree(d_row_ptr);
	hipFree(d_x);
	hipFree(d_y);

	for (int i = 0; i < N; i++)
		printf("y[%d] = %f\n", i, y[i]);

	return 0;
}