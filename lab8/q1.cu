#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>

__global__ void out(int* a, int* b, int wa) {
	int ridA = threadIdx.x;
	int ha = blockDim.x;
	
	int p = 1;
	if (ridA < ha) {
		for (int k = 0; k < wa; k++) {
			for (int z = 0; z <= ridA; z++) {
				p = p * a[ridA * wa + k];
			}
			b[ridA * wa + k] = p;
			p = 1;
		}
	}
}

int main() {
	int* a, * b, ha, wa;
	int* da, * db;

	printf("Enter the no.of rows and columns: ");
	scanf("%d %d", &ha, &wa);

	int size = sizeof(int) * ha * wa;

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	printf("Enter the matrix: ");
	for (int i = 0; i < ha; i++)
		for(int j = 0; j < wa; j++)
			scanf("%d", &a[i*wa + j]);

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);

	out << <1, ha >> > (da, db, wa);

	hipMemcpy(b, db, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < ha; i++) {
		for (int j = 0; j < wa; j++)
			printf("%d\t", b[i * wa + j]);
		printf("\n");
	}
	hipFree(da);
	hipFree(db);
	 
	return 0;
}